#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__
void kernel_print(int* input, int L){
    const int index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index < L){
        printf("%d = %d\n", index, input[index]);
    }
    
}


int main(int argc, char** argv) {

    int length = 8;

    thrust::host_vector<int> H1(length);
    thrust::device_vector<int> D1(length);

    thrust::sequence(D1.begin(), D1.end());

    thrust::device_vector<int> D2(length);
    D2 = D1;
    H1 = D1;

    int* D1_ptr = (int*)thrust::raw_pointer_cast(D1.data());

    for(int i = 0; i < length; i++){
        std::cout<<"D1="<<D1[i]<<std::endl;
    }
    kernel_print<<<32, 1>>>(D1_ptr, length);
    

    return 0;
}

